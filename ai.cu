#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include "ai.h"
#include "board.h"

namespace CUAI
{
    const int MaxDepth = 8;
    const int MaxBranch = 20;

    class cuai
    {
    public:
        hipDeviceProp_t devProp;
        const int BLOCK_PER_GRID = devProp.multiProcessorCount;
        const int THREAD_PER_BLOCK = devProp.maxThreadsPerBlock;
        const int TOTAL_COUNT = BLOCK_PER_GRID * THREAD_PER_BLOCK;

        cuai();
        ~cuai();
    } ai;

    cuai::cuai()
    {
        hipGetDeviceProperties(&devProp, 0);
    }

    cuai::~cuai()
    {
    }

    __host__ P solve(const ChessBoard &board)
    {
        ChessBoard brd = board;
        if (brd.state() == BoardState::Empty)
        {
            return P(7, 7);
        }
        else
        {
            PieceState clr = board.state() == BoardState::TurnOfBlack ? PieceState::Black : PieceState::White;

            std::memset(&WMat, 0, sizeof(WMat));
            assessBoard(brd, WMat);
            auto &&nextSteps = getNext(brd, WMat);
            WeightType best = -INF;
            WeightType A = -INF, B = INF;
            P res;

            for (const auto &i : nextSteps)
            {
                brd[i.first] = clr;
                best = std::max(best, DFS(brd, -B,
                                          -A, 0, PieceState(-clr)));

                brd[i.first] = PieceState::Empty;
                if (best > A)
                {
                    A = best;
                    res = i.first;
                }
            }
            return res;
        }
    }

} // namespace CUAI
